#include <stdio.h>
#include <stdlib.h>
#include <math.h>

extern "C" { // nvcc compiles in C++
#include <sparse_snp.h>
}

#include <omp.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

typedef unsigned int uint;

using namespace std;
using namespace hipcub;

