#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <snp_static.hpp> 

using namespace std;


/** Allocation */
SNP_static_ell::SNP_static_ell(uint n, uint m, int mode, bool debug) : SNP_model(n,m, mode, debug)
{
    //Allocate cpu variables
    this -> spiking_vector = (int*) malloc(sizeof(int)*m);
    memset(this->spiking_vector,0,  sizeof(int)*m);

    this->trans_matrix    = (int*)  malloc(sizeof(int)*n*m*2);
    memset(this->trans_matrix,-1,sizeof(int)*n*m*2);

    this->z_vector    = (int*) malloc(sizeof(int)*m);
    memset(this->z_vector,0,sizeof(int)*m);

    //Allocate device variables
    hipMalloc((&this->d_spiking_vector),  sizeof(int)*m);
    hipMemset(&this->d_spiking_vector, 0, sizeof(int)*m);
    //trans_matrix allocated when z is known

}

/** Free mem */
SNP_static_ell::~SNP_static_ell()
{
    free(this->spiking_vector);
    hipFree(this->d_spiking_vector);

    free(this->trans_matrix);
    hipFree(this->d_trans_matrix);

}

void SNP_static_ell::printTransMX(){
    for (int i=0; i<z; i++){
		
		for (int j=0; j<m; j++){
            int idx = (i*m*2 + j*2);
			std::cout << "(" << trans_matrix[idx] << "," << trans_matrix[idx+1] << ") ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";
}

void SNP_static_ell::include_synapse(uint i, uint j)
{
    // for (int r = rule_index[i]; r < rule_index[i+1]; r++) {
    //     trans_matrix[r*n+i] = -rules.c[r];
    //     if (j<n) trans_matrix[r*n+j] = rules.p[r];
    // }

    for (int r = rule_index[i]; r < rule_index[i+1]; r++) {
        //forgeting rules are not stored in trans_mx. 
        if(rules.p[r]>0){
            trans_matrix[z_vector[r]*m*2+r*2] = j;
            trans_matrix[(z_vector[r]*m*2+r*2)+1] = rules.p[r];
            z_vector[r]++;
        }
        
    }
}

void SNP_static_ell::init_compressed_matrix(){
    
    //find z (max output degree)
    // calc_z();
    // z++;

    // this->comp_trans_matrix    = (short*)  malloc(sizeof(short)*z*m*2);
    // memset(this->comp_trans_matrix,0,sizeof(short)*z*m*2);

    // //fill ell matrix
    // for(int i=0; i<m;i++){
    //     int aux_row=1; //start filling ell from position [1][rule]. row 0 reserved for (negative) c values.
    //     for(int j=0; j<n;j++){
    //         short num = trans_matrix[i*n+j]; //[i][j]
    //         if(num!=0){
    //             if(num>0){
    //                 int idx = (m*aux_row+i)*2;
    //                 comp_trans_matrix[idx] = j;
    //                 comp_trans_matrix[idx+1] = num;
                    
                    
    //                 aux_row++;

    //             }else{
    //                 //first row
    //                 comp_trans_matrix[i*2] = j;
    //                 comp_trans_matrix[i*2+1] = num;
    //                 // std::cout << num << " " << "i:" << i << " j:" <<j << " m:" <<m <<" aux_row:" <<aux_row <<"\n";
    //             }

    //         }
    //     }
        

    // }
    
    //get z (num of rows of the mx) max(z_vector)
    for(int r=0; r<m; r++){
        int aux_z=z_vector[r];
        if(aux_z>z){
            z=aux_z;
        }
    }


    // this -> trans_matrix = (short *) realloc(trans_matrix, z*m*2);

}

void SNP_static_ell::load_transition_matrix () 
{
    //handled by sublcasses
    init_compressed_matrix();

    hipMalloc((&this->d_trans_matrix),  sizeof(int)*z*m*2);
    hipMemcpy(d_trans_matrix,  trans_matrix,   sizeof(int)*z*m*2,  hipMemcpyHostToDevice); 

    // TODO The following should be done in another function, but for simplicity I put it here
    // TODO check if we need to set matrices for spiking and configuration vectors
}

__global__ void kalc_spiking_vector_ell(int* spiking_vector, int* delays_vector, int* conf_vector, int* rule_index, uint* rnid, int* rc, int* rei, int* ren, uint* rd, uint n)
{
    uint nid = threadIdx.x+blockIdx.x*blockDim.x;
    if (nid<n && delays_vector[nid]==0) {
        //vector<int> active_rule_idxs_ni;
        for (int r=rule_index[nid]; r<rule_index[nid+1]; r++){
            uchar i = rei[r];
            uchar n = ren[r];
            int x = conf_vector[nid];
            if ((int) (i&(x==n)) || ((1-i)&(x>=n))){
                conf_vector[nid]-= rc[r];
                spiking_vector[r] = 1;
                delays_vector[nid] = rd[r]; 
                break;
            }
        }
        
    }
}

void SNP_static_ell::calc_spiking_vector() 
{
    uint bs = 256;
    uint gs = (n+255)/256;
    
    kalc_spiking_vector_ell<<<gs,bs>>>(d_spiking_vector, d_delays_vector, d_conf_vector, d_rule_index, d_rules.nid, d_rules.c, d_rules.Ei, d_rules.En, d_rules.d, n);
    hipDeviceSynchronize();

    hipMemcpy(spiking_vector, d_spiking_vector,  sizeof(int)*m, hipMemcpyDeviceToHost);
    hipMemcpy(delays_vector, d_delays_vector,  sizeof(int)*n, hipMemcpyDeviceToHost);

}


__global__ void kalc_transition_ell(int* spiking_vector, int* trans_matrix, int* conf_vector, int * delays_vector, uint* rnid, int z, int m){
    int rid = threadIdx.x+blockIdx.x*blockDim.x;
    
    //nid<n
    
    if (rid<m && spiking_vector[rid]>0 && delays_vector[rnid[rid]]==0){
        spiking_vector[rid] = 0;
        for(int i=0; i<z; i++){
            int neuron = trans_matrix[m*2*i+rid*2];
            int value = trans_matrix[m*2*i+rid*2+1];
            if(neuron==-1 && value==-1){
                break;
            }
            if(delays_vector[neuron]==0){
                atomicAdd((int *)&conf_vector[neuron], (int)value);
            }
            
        }
        
        // printf("%d ",conf_vector[nid]);
    }
    



}

__global__ void update_delays_vector(int * delays_vector, int n){
    int nid=threadIdx.x+blockIdx.x*blockDim.x;
    if(nid<n && delays_vector[nid]>0){
        delays_vector[nid]--;
    }
}

void SNP_static_ell::calc_transition()
{
    kalc_transition_ell<<<n+255,256>>>(d_spiking_vector,d_trans_matrix, d_conf_vector, d_delays_vector, d_rules.nid,z,m);
    hipDeviceSynchronize();
    update_delays_vector<<<n+255,256>>>(d_delays_vector, n);
    hipDeviceSynchronize();

}

