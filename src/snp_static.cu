#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
//#define assert
//#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#define IDXCB(i,j,ld) (((j)*(ld))+(i))   // indexing for CUBLAS

#include <snp_static.hpp>

using namespace std;

/** Allocation */
SNP_static_cublas::SNP_static_cublas(uint n, uint m) : SNP_model(n,m)
{
    // n is num of rows, m is num of colums. 
    hipError_t cudaStat;
    hipblasStatus_t stat;
    // done by subclasses
    this->trans_matrix    = (short*)  malloc(sizeof(short)*n*m);
    memset(this->trans_matrix,0,sizeof(short)*n*m);
    /*for (int i = 0; i < m; i++) // for each row = rule
        for (int j = 0; j<n; j++) // for each column = neuron
            this->trans_matrix[i*n+j] = 0;*/

    cudaStat  = hipMalloc(&this->d_trans_matrix,  sizeof(short)*n*m);
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        exit(1);
        // return EXIT_FAILURE;
    }

    this->cublas_handle = (hipblasHandle_t *) malloc(sizeof(hipblasHandle_t));
    stat = hipblasCreate((hipblasHandle_t *)cublas_handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        //return EXIT_FAILURE;
        exit(1);
    }
}

/** Free mem */
SNP_static_cublas::~SNP_static_cublas()
{
    free(this->trans_matrix);
    free(this->cublas_handle);
    hipFree(this->d_trans_matrix);
}

void SNP_static_cublas::include_synapse(uint i, uint j)
{
    for (int r = rule_index[i]; r < rule_index[i+1]; r++) {
        //trans_matrix[r*n+i] = rules.c[r];
        //trans_matrix[r*n+j] = rules.p[r];
        trans_matrix[IDXCB(r,i,n)] = rules.c[r];
        trans_matrix[IDXCB(r,i,n)] = rules.p[r];
    }
}


void SNP_static_cublas::load_transition_matrix () 
{
    //handled by sublcasses
    //hipMemcpy(d_trans_matrix,  trans_matrix,   sizeof(short)*n*m,  hipMemcpyHostToDevice); // now handled by CuBLAS
    hipblasStatus_t stat;
    stat = hipblasSetMatrix (n, m, sizeof(*this->trans_matrix), this->trans_matrix, n, this->d_trans_matrix, n);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("loading transition matrix to GPU failed");
        exit(1);
        // return EXIT_FAILURE;
    }

    // TODO The following shoud be done in another function, but for simplicity I put it here
    // TODO check if we need to set matrices for spiking and configuration vectors
}


/*__global__ void ksmvv (short* a, short* v, short* w, uint m) i
{
    uint n = blockIdx.x;
    uint acum = =0;
    for (uint i=tid; i<m; i+=blockDim.x) {
        acum+=a[i]*v[i];
    }
    __syncthreads();

    // reduce

    if (threadIdx.x==0)
        w[n] = acum;
}*/

void SNP_static_cublas::calc_transition()
{
    hipblasStatus_t stat;
    float alpha =1.0f;
    float beta =0.0f;
    stat = hipblasSgemv((hipblasHandle_t)*this->cublas_handle,HIPBLAS_OP_N,n,m,&alpha,this->d_trans_matrix,n,this->d_spiking_vector,1,&beta,this->d_conf_vector,1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("matrix-vector multiplication on GPU failed");
        return EXIT_FAILURE;
    }
}

