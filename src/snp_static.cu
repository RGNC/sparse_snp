#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include <snp_static.hpp> 

using namespace std;

/** Allocation */
SNP_static::SNP_static(uint n, uint m) : SNP_model(n,m)
{
    //Allocate cpu variables
    this -> spiking_vector = (ushort*) malloc(sizeof(ushort)*m);
    memset(this->spiking_vector,0,  sizeof(ushort)*m);

    this->trans_matrix    = (short*)  malloc(sizeof(short)*n*m);
    memset(this->trans_matrix,0,sizeof(short)*n*m);

    //Allocate device variables
    hipMalloc((&this->d_spiking_vector),  sizeof(ushort)*m);
    hipMalloc((&this->d_trans_matrix),  sizeof(short)*n*m);
    
}

/** Free mem */
SNP_static::~SNP_static()
{
    free(this->spiking_vector);
    hipFree(this->d_spiking_vector);

    free(this->trans_matrix);
    hipFree(this->d_trans_matrix);
}

void SNP_static::printTransMX()
{
    for (int i=0; i<m; i++){
		
		for (int j=0; j<n; j++){
			std::cout << trans_matrix[i*n + j] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";
}

void SNP_static::include_synapse(uint i, uint j)
{
    //store by columns for better VxM performance
    for (int r = rule_index[i]; r < rule_index[i+1]; r++) {
        trans_matrix[i*m+r] = 0;  
        trans_matrix[j*m+r] = rules.p[r];
    }
}


void SNP_static::load_transition_matrix () 
{

    hipMemcpy(d_trans_matrix,  trans_matrix,   sizeof(short)*n*m,  hipMemcpyHostToDevice); 

    // TODO The following should be done in another function, but for simplicity I put it here
    // TODO check if we need to set matrices for spiking and configuration vectors
}




/*__global__ void ksmvv (short* a, short* v, short* w, uint m) i
{
    uint n = blockIdx.x;
    uint acum = =0;
    for (uint i=tid; i<m; i+=blockDim.x) {
        acum+=a[i]*v[i];
    }
    __syncthreads();

    // reduce

    if (threadIdx.x==0)
        w[n] = acum;
}*/
__global__ void kalc_spiking_vector(ushort* spiking_vector, int* delays_vector, ushort* rd, int* conf_vector, int* rule_index,short* rc, short* rei, short* ren, uint n)
{
    uint nid = threadIdx.x+blockIdx.x*blockDim.x;

    if (nid<n && delays_vector[nid]==0) {

        
        for (uint r=rule_index[nid]; r<rule_index[nid+1]; r++){

            uchar e_i = rei[r];
            uchar e_n = ren[r];
            int x = conf_vector[nid];

            if ((int) (e_i&(x==e_n)) || ((1-e_i)&(x>=e_n))) {
                
                spiking_vector[r] = 1;
                conf_vector[nid]-=rc[r];
                delays_vector[nid] = rd[r];

                break;
            }

            

        }

        
    }
}

void SNP_static::calc_spiking_vector() 
{
    uint bs = 256;
    uint gs = (n+255)/256;
    kalc_spiking_vector<<<gs,bs>>>(d_spiking_vector, d_delays_vector, d_rules.d, d_conf_vector, d_rule_index,d_rules.c, d_rules.Ei, d_rules.En, n);
    hipDeviceSynchronize();

    //send spiking_vector and delays_vector to host in order to decide if stop criterion has been reached
    hipMemcpy(spiking_vector, d_spiking_vector,  sizeof(ushort)*m, hipMemcpyDeviceToHost);
    hipMemcpy(delays_vector, d_delays_vector,  sizeof(int)*n, hipMemcpyDeviceToHost);


}

__global__ void kalc_transition(ushort* spiking_vector, short* trans_matrix, int* conf_vector,int * delays_vector, uint * rnid , int n, int m){
    int nid = threadIdx.x+blockIdx.x*blockDim.x;
    //nid<n
    if (nid<n && delays_vector[nid]==0){
        for (int r=0; r<m; r++){
            //only sum spikes from neurons that are open, even though spiking_vector[r]=1. TODO: In cublas make trans_matrix_copy and make 0 every row of every rule corresponding to a closed neuron.
            if(delays_vector[rnid[r]] == 0){
                conf_vector[nid] += spiking_vector[r] * trans_matrix[nid*m+r]; 
                spiking_vector[r] = 0; //disable rule that has been used
            }
            
        }



        // printf("%d ",conf_vector[nid]);
    }

    if(nid<n && delays_vector[nid]>0){
        delays_vector[nid]--;
    }

}

void SNP_static::calc_transition()
{
    kalc_transition<<<n+255,256>>>(d_spiking_vector,d_trans_matrix, d_conf_vector, d_delays_vector, d_rules.nid,n,m);
    hipDeviceSynchronize();

}

