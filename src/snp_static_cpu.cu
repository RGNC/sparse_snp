#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include <snp_static.hpp> //"../include/snp_static.hpp" // <snp_static.hpp> 

using namespace std;

/** Allocation */
SNP_static_cpu::SNP_static_cpu(uint n, uint m) : SNP_model_cpu(n,m)
{
    // n is num of rows, m is num of colums. 
    
    // done by subclasses
    this->trans_matrix    = (short*)  malloc(sizeof(short)*n*m);
    memset(this->trans_matrix,0,sizeof(short)*n*m);


}

/** Free mem */
SNP_static_cpu::~SNP_static_cpu()
{
    free(this->trans_matrix);
}

void SNP_static_cpu::include_synapse(uint i, uint j)
{
    for (int r = rule_index[i]; r < rule_index[i+1]; r++) {
        trans_matrix[r*n+i] = -rules.c[r];
        if (j<n) trans_matrix[r*n+j] = rules.p[r];
    }
}


void SNP_static_cpu::calc_transition()
{
    for (int nid=0; nid<n; nid++){
        for (int r=0; r<m; r++){
            conf_vector[nid] += spiking_vector[r] * trans_matrix[r*n+nid];
        }
        
    }
    
}

